#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"

#define LENGTH 100000
#define WIDTH 100
#define HEIGHT 100

float _max(float a, float b);
float _abs(float f);
__global__ void add_vecs(const int len, const float* a, const float* b, float* c);

int main(int argc, char** argv)
{
	srand(time(NULL));

	float* x = (float*)malloc(sizeof(float) * LENGTH);
	float* y = (float*)malloc(sizeof(float) * LENGTH);
	float* z = (float*)malloc(sizeof(float) * LENGTH);
	float* zp = (float*)malloc(sizeof(float) * LENGTH);

	for(int i = 0;i < LENGTH;i++)
	{
		x[i] = rand() / ((float)rand());
		y[i] = rand() / ((float)rand());
		z[i] = 0.0f;

		zp[i] = x[i] + y[i];
	}

	float* d_x;
	float* d_y;
	float* d_z;
	
	hipMalloc<float>(&d_x,sizeof(float) * LENGTH);
	hipMalloc<float>(&d_y,sizeof(float) * LENGTH);
	hipMalloc<float>(&d_z,sizeof(float) * LENGTH);

	hipMemcpy(d_x,x,LENGTH,hipMemcpyHostToDevice);
	hipMemcpy(d_y,y,LENGTH,hipMemcpyHostToDevice);
	hipMemcpy(d_z,z,LENGTH,hipMemcpyHostToDevice);

	dim3 blocks(LENGTH / 256 + 1);
	dim3 threads(256);
	add_vecs<<<blocks,threads>>>(LENGTH,d_x,d_y,d_z);

	hipMemcpy(z,d_z,LENGTH,hipMemcpyDeviceToHost);

	float max_err = 0.0f;

	for(int i = 0;i < LENGTH;i++)
		max_err = _max(max_err,_abs(z[i] - zp[i]));

	printf("Maximum error is: %.6f",max_err);
	
	free(x);
	free(y);
	free(z);
	
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);

	return 0;
}

float _max(float a, float b)
{return a > b ? a : b;}

float _abs(float f)
{return f < 0.0f ? -f : f;}

__global__ void add_vecs(const int len, const float* a, const float* b, float* c)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(index < len)
		c[index] = a[index] + b[index];

	return;
}