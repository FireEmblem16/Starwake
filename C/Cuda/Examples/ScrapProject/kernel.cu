#include <stdlib.h>
#include "hip/hip_runtime.h"

#define LENGTH 100000

__global__ void copy_vec(int len, const float* a, float* b);

int main(int argc, char** argv)
{
	float* x;
	float* y;

	x = (float*)malloc(LENGTH * sizeof(float));
	y = (float*)malloc(LENGTH * sizeof(float));


	for(int i = 0;i < LENGTH;i++)
	{
		x[i] = rand();
		y[i] = 0.0f;
	}

	float* d_x;
	float* d_y;

	hipMalloc(&d_x,LENGTH * sizeof(float));
	hipMalloc(&d_y,LENGTH * sizeof(float));

	hipMemcpy(d_x,x,LENGTH * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_y,y,LENGTH * sizeof(float),hipMemcpyHostToDevice);

	dim3 blocks(LENGTH / 256 + 1);
	dim3 threads(256);

	copy_vec<<<blocks,threads>>>(LENGTH,d_x,d_y);
	
	hipMemcpy(y,d_y,LENGTH * sizeof(float),hipMemcpyDeviceToHost);

	free(x);
	free(y);
	
	hipFree(d_x);
	hipFree(d_y);

	return 0;
}

__global__ void copy_vec(int len, const float* a, float* b)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index < len)
		b[index] = 4.0f * a[index] / 2.415f;

	return;
}